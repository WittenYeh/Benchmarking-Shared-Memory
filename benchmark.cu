#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void writeSharedMemorySame() {
    __shared__ int shared_data[256];
    shared_data[0] = -1;
    // 所有线程写入同一位置
    if (shared_data[0] < 0) {
        shared_data[0] = threadIdx.x;
    }
    __syncthreads(); // 确保所有线程完成写入
}

__global__ void writeSharedMemoryDifferent() {
    __shared__ int shared_data[256]; // 假设最大线程数为256

    // 每个线程写入不同位置
    shared_data[threadIdx.x] = threadIdx.x;
    __syncthreads(); // 确保所有线程完成写入
}

int main() {
    const int blockSize = 256; // 线程块大小
    const int numBlocks = 1 << 20;   // 线程块数量

    // 记录时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 测试不同位置
    hipEventRecord(start);
    for (int i = 0; i < 1 << 26; ++i)
        writeSharedMemoryDifferent<<<numBlocks, blockSize>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTimeDifferent;
    hipEventElapsedTime(&elapsedTimeDifferent, start, stop);

    // 测试相同位置
    hipEventRecord(start);
    for (int i = 0; i < 1 << 26; ++i)
        writeSharedMemorySame<<<numBlocks, blockSize>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTimeSame;
    hipEventElapsedTime(&elapsedTimeSame, start, stop);

    // 输出结果
    printf("Time for same position: %f ms\n", elapsedTimeSame);
    printf("Time for different positions: %f ms\n", elapsedTimeDifferent);

    // 清理资源
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
